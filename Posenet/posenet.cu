#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;

// Parameters
const int H = 224;
const int W = 224;
const int C_in = 16;
const int reduced_C = 256;

// ---------- CUDA Kernels ----------

// 1x1 Conv
__global__ void conv1x1_kernel(float* input, float* output, float* weights, int H, int W, int C_in, int C_out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) return;

    for (int c_out = 0; c_out < C_out; ++c_out) {
        float sum = 0.0f;
        for (int c_in = 0; c_in < C_in; ++c_in) {
            int idx = (c_in * H * W) + (y * W) + x;
            int weight_idx = (c_out * C_in) + c_in;
            sum += input[idx] * weights[weight_idx];
        }
        output[(c_out * H * W) + (y * W) + x] = fmaxf(0.0f, sum); // ReLU
    }
}

// 3x3 Conv
__global__ void conv3x3_kernel(float* input, float* output, float* kernel, int H, int W, int C_in, int K, int C_out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int pad = K / 2;
    if (x >= W || y >= H) return;

    for (int c_out = 0; c_out < C_out; ++c_out) {
        float sum = 0.0f;
        for (int c_in = 0; c_in < C_in; ++c_in) {
            for (int i = 0; i < K; ++i) {
                for (int j = 0; j < K; ++j) {
                    int xi = x + i - pad;
                    int yj = y + j - pad;
                    if (xi >= 0 && xi < W && yj >= 0 && yj < H) {
                        int idx = (c_in * H * W) + (yj * W) + xi;
                        sum += input[idx] * kernel[((c_out * C_in + c_in) * K * K) + (i * K) + j];
                    }
                }
            }
        }
        output[(c_out * H * W) + (y * W) + x] = fmaxf(0.0f, sum); // ReLU
    }
}

// Final 1x1 Conv (Output 6 channels)
__global__ void conv1x1_final_kernel(float* input, float* output, float* weights, int H, int W, int C_in, int C_out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) return;

    for (int c_out = 0; c_out < C_out; ++c_out) {
        float sum = 0.0f;
        for (int c_in = 0; c_in < C_in; ++c_in) {
            int idx = (c_in * H * W) + (y * W) + x;
            int weight_idx = (c_out * C_in) + c_in;
            sum += input[idx] * weights[weight_idx];
        }
        output[(c_out * H * W) + (y * W) + x] = sum;
    }
}

// Global Avg Pool
__global__ void global_avg_pool_kernel(float* input, float* output, int H, int W, int C_in) {
    int c = threadIdx.x;
    if (c >= C_in) return;
    float sum = 0.0f;
    for (int y = 0; y < H; ++y)
        for (int x = 0; x < W; ++x)
            sum += input[(c * H * W) + (y * W) + x];
    output[c] = sum / (H * W);
}

// ---------- Helper Functions ----------

void load_feature_from_csv(const string& csv_path, vector<float>& feature) {
    ifstream file(csv_path);
    string line;
    int idx = 0;
    while (getline(file, line)) {
        size_t pos = 0;
        while ((pos = line.find(',')) != string::npos) {
            feature[idx++] = stof(line.substr(0, pos));
            line.erase(0, pos + 1);
        }
        feature[idx++] = stof(line); // last val
    }
}

void save_pose_to_csv(const vector<float>& pose, const string& output_path) {
    ofstream out_csv(output_path);
    for (int i = 0; i < pose.size(); ++i) {
        out_csv << pose[i];
        if (i != pose.size() - 1) out_csv << ",";
    }
    out_csv << endl;
    out_csv.close();
}

// ---------- Main Function ----------

int main(int argc, char** argv) {
    if (argc < 4) {
        cout << "Usage: " << argv[0] << " <feature_csv_img1> <feature_csv_img2> <output_pose_csv>" << endl;
        return -1;
    }

    string feature_csv1 = argv[1];
    string feature_csv2 = argv[2];
    string output_pose = argv[3];

    // Step 1: Load Features
    vector<float> feature1(C_in * H * W);
    vector<float> feature2(C_in * H * W);
    load_feature_from_csv(feature_csv1, feature1);
    load_feature_from_csv(feature_csv2, feature2);

    // Step 2: Allocate GPU memory
    float *d_feat1, *d_feat2, *d_feat1_reduced, *d_feat2_reduced;
    hipMalloc(&d_feat1, C_in * H * W * sizeof(float));
    hipMalloc(&d_feat2, C_in * H * W * sizeof(float));
    hipMalloc(&d_feat1_reduced, reduced_C * H * W * sizeof(float));
    hipMalloc(&d_feat2_reduced, reduced_C * H * W * sizeof(float));

    hipMemcpy(d_feat1, feature1.data(), C_in * H * W * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_feat2, feature2.data(), C_in * H * W * sizeof(float), hipMemcpyHostToDevice);

    // Step 3: 1x1 Convs
    vector<float> conv1x1_weights(reduced_C * C_in);
    for (auto& w : conv1x1_weights) w = static_cast<float>(rand()) / RAND_MAX;
    float* d_conv1x1_weights;
    hipMalloc(&d_conv1x1_weights, reduced_C * C_in * sizeof(float));
    hipMemcpy(d_conv1x1_weights, conv1x1_weights.data(), reduced_C * C_in * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((W + block.x - 1) / block.x, (H + block.y - 1) / block.y);
    conv1x1_kernel<<<grid, block>>>(d_feat1, d_feat1_reduced, d_conv1x1_weights, H, W, C_in, reduced_C);
    conv1x1_kernel<<<grid, block>>>(d_feat2, d_feat2_reduced, d_conv1x1_weights, H, W, C_in, reduced_C);
    hipDeviceSynchronize();

    // Step 4: Concatenate
    float* d_concat;
    hipMalloc(&d_concat, 2 * reduced_C * H * W * sizeof(float));
    hipMemcpy(d_concat, d_feat1_reduced, reduced_C * H * W * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(d_concat + reduced_C * H * W, d_feat2_reduced, reduced_C * H * W * sizeof(float), hipMemcpyDeviceToDevice);

    // Step 5: 3x3 Conv
    const int conv_out_C = 256;
    float* d_conv1_out;
    hipMalloc(&d_conv1_out, conv_out_C * H * W * sizeof(float));
    vector<float> conv3x3_weights(conv_out_C * 2 * reduced_C * 3 * 3);
    for (auto& w : conv3x3_weights) w = static_cast<float>(rand()) / RAND_MAX;
    float* d_conv3x3_weights;
    hipMalloc(&d_conv3x3_weights, conv3x3_weights.size() * sizeof(float));
    hipMemcpy(d_conv3x3_weights, conv3x3_weights.data(), conv3x3_weights.size() * sizeof(float), hipMemcpyHostToDevice);
    conv3x3_kernel<<<grid, block>>>(d_concat, d_conv1_out, d_conv3x3_weights, H, W, 2 * reduced_C, 3, conv_out_C);
    hipDeviceSynchronize();

    // Step 6: Final 1x1 Conv → 6D Pose
    const int final_out_C = 6;
    float* d_final_out;
    hipMalloc(&d_final_out, final_out_C * H * W * sizeof(float));
    vector<float> conv1x1_final_weights(final_out_C * conv_out_C);
    for (auto& w : conv1x1_final_weights) w = static_cast<float>(rand()) / RAND_MAX;
    float* d_conv1x1_final_weights;
    hipMalloc(&d_conv1x1_final_weights, final_out_C * conv_out_C * sizeof(float));
    hipMemcpy(d_conv1x1_final_weights, conv1x1_final_weights.data(), final_out_C * conv_out_C * sizeof(float), hipMemcpyHostToDevice);
    conv1x1_final_kernel<<<grid, block>>>(d_conv1_out, d_final_out, d_conv1x1_final_weights, H, W, conv_out_C, final_out_C);
    hipDeviceSynchronize();

    // Step 7: Global Avg Pool → Final Pose Vector
    float* d_pose;
    hipMalloc(&d_pose, final_out_C * sizeof(float));
    global_avg_pool_kernel<<<1, final_out_C>>>(d_final_out, d_pose, H, W, final_out_C);
    hipDeviceSynchronize();

    // Step 8: Copy back & Save
    vector<float> pose(final_out_C);
    hipMemcpy(pose.data(), d_pose, final_out_C * sizeof(float), hipMemcpyDeviceToHost);
    save_pose_to_csv(pose, output_pose);

    cout << "Predicted 6D Pose saved to: " << output_pose << endl;

    // Cleanup
    hipFree(d_feat1); hipFree(d_feat2);
    hipFree(d_feat1_reduced); hipFree(d_feat2_reduced);
    hipFree(d_conv1x1_weights); hipFree(d_concat);
    hipFree(d_conv1_out); hipFree(d_conv3x3_weights);
    hipFree(d_final_out); hipFree(d_conv1x1_final_weights);
    hipFree(d_pose);

    return 0;
}
